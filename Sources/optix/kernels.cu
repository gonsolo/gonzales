#include "hip/hip_runtime.h"
#include <stdio.h>
#include <cstdint>
#include "optix_device.h"
#include "LaunchParameters.h"

extern "C" __constant__ LaunchParameters launchParameters;

extern "C" __global__ void __closesthit__radiance() {}
extern "C" __global__ void __anyhit__radiance() {}
extern "C" __global__ void __miss__radiance() {}
extern "C" __global__ void __raygen__renderFrame() {
	const int x = optixGetLaunchIndex().x;
	const int y = optixGetLaunchIndex().y;
	//if (x == 0 && y  == 0) {
	//	printf("Render frame kernel, x, y: %i %i!\n", x, y);
	//}

	const uint8_t r = x * 16;
	const uint8_t g = 0;
	const uint8_t b = 0;
	const uint8_t a = 255;
	const int components = 4;
	const int index = y * launchParameters.width * components + x * components;
	uint8_t* p = (uint8_t*)launchParameters.pointerToPixels;
	p[index + 0] = r;
	p[index + 1] = g;
	p[index + 2] = b;
	p[index + 3] = a;
}
